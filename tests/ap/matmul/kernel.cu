// auto generated

#include "epilogue_op.h"

// template <typename T>
// struct EpilogueArguments {
//   typename ap::ScaleFunctor<T>::Arguments scale_args;
// };
 
template <typename T>
struct EpilogueFunctor {
  using Arguments = typename ap::ScaleFunctor<T>::Arguments;

  __forceinline__ __host__ __device__
  T operator()(T x, Arguments args) const {
    return ap::ScaleFunctor<T>()(x, args);
  }
};

#include "cutlass_matmul.cuh"

extern "C" {

void MatmulAddUnaryKernel(hipStream_t stream, const void* input, const void* weight, const void* bias, void* output, int m, int n, int k) {
  GemmEpilogueParams params;

  params.m = m;
  params.n = n;
  params.k = k;

  params.input = input;
  params.weight = weight;
  params.bias = bias;
  params.output = output;

  params.stream = stream;

  ap::ScaleFunctor<float>::Arguments unary_args{1.0};
  CutlassMatmulAddUnary<cutlass::half_t, float, EpilogueFunctor>(params, unary_args);
}

void MatmulAddBinaryKernel(hipStream_t stream, const void* input, const void* weight, const void* bias, void* broadcast, void* broadcast_out, void* output, int m, int n, int k) {
  GemmBroadcastEpilogueParams params;

  params.m = m;
  params.n = n;
  params.k = k;

  params.input = input;
  params.weight = weight;
  params.bias = bias;
  params.output = output;

  params.broadcast = broadcast;
  params.broadcast_out = broadcast_out;

  params.stream = stream;

  CutlassMatmulAddBinary<cutlass::half_t, float>(params);
}

}
