#include "native_matmul.cuh"

extern "C" {

void NativeMatmulAddKernel(hipStream_t* stream, const void* input, const void* weight, const void* bias, void* output, int batch_count, int m, int n, int k, bool transpose_b) {
  ap::GemmEpilogueParams params;

  params.batch_count = batch_count;
  params.m = m;
  params.n = n;
  params.k = k;

  params.input = input;
  params.weight = weight;
  params.bias = bias;
  params.output = output;

  params.stream = *stream;
  native::MatmulAdd<float>(params);
}

}
